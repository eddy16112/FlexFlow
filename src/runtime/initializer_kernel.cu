/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "initializer.h"
#include "accessor.h"
#include "model.h"
#include "cuda_helper.h"
#include <ctime>

void UniformInitializer::init_task(const Task* task,
                                   const std::vector<PhysicalRegion>& regions,
                                   Context ctx, Runtime* runtime)
{
  assert(regions.size() == 1);
  assert(task->regions.size() == 1);
  TensorAccessorW<float, 2> accW(regions[0], task->regions[0],
      FID_DATA, ctx, runtime, false/*readOutput*/);
  int inputDim = accW.rect.hi[0] - accW.rect.lo[0] + 1;
  int outputDim = accW.rect.hi[1] - accW.rect.lo[1] + 1;
  UniformInitializer* initializer = (UniformInitializer*) task->args;
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  //fprintf(stderr, "seed = %d\n", initializer->seed);
  hiprandSetPseudoRandomGeneratorSeed(gen, initializer->seed);
  checkCUDA(hiprandGenerateUniform(gen, accW.ptr, accW.rect.volume()));
  scale_kernel<<<GET_BLOCKS(accW.rect.volume()), CUDA_NUM_THREADS>>>(
      accW.ptr, accW.rect.volume(), initializer->min_val, initializer->max_val);
  checkCUDA(hipDeviceSynchronize());
  hiprandDestroyGenerator(gen);
}

void NormInitializer::init_task(const Task* task,
                                const std::vector<PhysicalRegion>& regions,
                                Context ctx, Runtime* runtime)
{
  assert(regions.size() == 1);
  assert(task->regions.size() == 1);
  Domain domain = runtime->get_index_space_domain(
      ctx, task->regions[0].region.get_index_space());
  float* w;
  switch (domain.get_dim()) {
    case 1:
    {
      TensorAccessorW<float, 1> accW(regions[0], task->regions[0],
          FID_DATA, ctx, runtime, false/*readOutput*/);
      w = accW.ptr;
      break;
    }
    case 2:
    {
      TensorAccessorW<float, 2> accW(regions[0], task->regions[0],
          FID_DATA, ctx, runtime, false/*readOutput*/);
      w = accW.ptr;
      break;
    }
    case 3:
    {
      TensorAccessorW<float, 3> accW(regions[0], task->regions[0],
          FID_DATA, ctx, runtime, false/*readOutput*/);
      w = accW.ptr;
      break;
    }
    default:
      assert(false);
  }
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  NormInitializer* initializer = (NormInitializer*) task->args;
  //fprintf(stderr, "seed = %d\n", initializer->seed);
  hiprandSetPseudoRandomGeneratorSeed(gen, initializer->seed);
  //fprintf(stderr, "domain.volume() = %zu mean(%.4lf) var(%.4lf)\n",
  //    domain.get_volume(), initializer->mean, initializer->stddev);
  checkCUDA(hiprandGenerateNormal(gen, w, domain.get_volume(),
      initializer->mean, initializer->stddev));
  checkCUDA(hipDeviceSynchronize());
  hiprandDestroyGenerator(gen);
}

void ZeroInitializer::init_task(const Task* task,
                                const std::vector<PhysicalRegion>& regions,
                                Context ctx, Runtime* runtime)
{
  assert(regions.size() == task->regions.size());
  for (size_t i = 0; i < regions.size(); i++) {
    Domain domain = runtime->get_index_space_domain(
        ctx, task->regions[i].region.get_index_space());
    float* w;
    switch (domain.get_dim()) {
      case 0:
      {
        // Do not support 0-dim parameters
        assert(false);
        break;
      }
      case 1:
      {
        TensorAccessorW<float, 1> accW(
            regions[i], task->regions[i], FID_DATA, ctx, runtime, false/*readOutput*/);
        w = accW.ptr;
        break;
      }
      case 2:
      {
        TensorAccessorW<float, 2> accW(
            regions[i], task->regions[i], FID_DATA, ctx, runtime, false/*readOutput*/);
        w = accW.ptr;
        break;
      }
      case 3:
      {
        TensorAccessorW<float, 3> accW(
            regions[i], task->regions[i], FID_DATA, ctx, runtime, false/*readOutput*/);
        w = accW.ptr;
        break;
      }
      default:
      {
         assert(false);
         break;
      }
    }
    assign_kernel<<<GET_BLOCKS(domain.get_volume()), CUDA_NUM_THREADS>>>(
        w, domain.get_volume(), 0.0f);
  }
  checkCUDA(hipDeviceSynchronize());
}
