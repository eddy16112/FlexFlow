#include "hip/hip_runtime.h"
/* Copyright 2017 Stanford, NVIDIA
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::batch_norm(std::string name, Tensor input, bool relu)
{
  assert(input.numDim == 4); //Only support 4D BN for now
  IndexSpaceT<4> task_is;
  BatchNorm *bn = new BatchNorm(name, config, input, task_is, relu);
  layers.push_back(bn);
  return bn->output;
}

/*
  locals[0] = scale
  locals[1] = bias
*/
BatchNorm::BatchNorm(std::string _name, FFConfig _config,
                     Tensor _input, IndexSpaceT<4> _task_is,
                     bool _relu)
: Op(_name, _input), relu(_relu), profiling(_config.profiling)
{
  Context ctx = _config.lg_ctx;
  HighLevelRuntime* runtime = _config.lg_hlr;
  Rect<4> part_rect = runtime->get_index_space_domain(ctx, task_is);
  num_replica = part_rect.volume();
  // Create output tensor
  int output_w = _input.adim[0];
  int output_h = _input.adim[1];
  int output_c = _input.adim[2];
  int output_n = _input.adim[3];
  int num_par_w = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_h = part_rect.hi[1] - part_rect.lo[1] + 1;
  int num_par_c = part_rect.hi[2] - part_rect.lo[2] + 1;
  int num_par_n = part_rect.hi[3] - part_rect.lo[3] + 1;

  FieldSpace fs = _config.field_space;
  Rect<4> output_rect(Point<4>(0, 0, 0, 0),
      Point<4>(output_w-1, output_h-1, output_c-1, output_n-1));
  IndexSpaceT<4> output_is = runtime->create_index_space(ctx, output_rect);
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  LogicalRegion output_grad_lr = runtime->create_logical_region(ctx, output_is, fs);
  int extent_w = (output_w + num_par_w - 1) / num_par_w;
  int extent_h = (output_h + num_par_h - 1) / num_par_h;
  int extent_c = output_c / num_par_c;
  int extent_n = output_n / num_par_n;
  assert(output_c % num_par_c == 0);
  assert(output_n % num_par_n == 0);
  Rect<4> ext(Point<4>(0, 0, 0, 0),
      Point<4>(extent_w-1, extent_h-1, extent_c-1, extent_n-1));
  Transform<4, 4, coord_t> trans;
  for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
      trans[i][j] = 0;
  trans[0][0] = extent_w;
  trans[1][1] = extent_h;
  trans[2][2] = extent_c;
  trans[3][3] = extent_n;
  IndexPartition output_ip =
    runtime->create_partition_by_restriction(ctx, output_is, task_is, trans, ext);
  assert(runtime->is_index_partition_disjoint(ctx, output_ip));
  assert(runtime->is_index_partition_complete(ctx, output_ip));
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  LogicalPartition output_grad_lp =
    runtime->get_logical_partition(ctx, output_grad_lr, output_ip);

  int bias_nc = num_replica * _input.adim[2]; /*input_channels*/
  Rect<1, coord_t> bias_grad_rect(0, bias_nc - 1);
  Rect<1, coord_t> bias_rect(0, _input.adim[2] - 1);
  IndexSpaceT<1> bias_is = runtime->create_index_space(ctx, bias_rect);
  IndexSpaceT<1> bias_grad_is = runtime->create_index_space(ctx, bias_grad_rect);
  LogicalRegion bias_lr = runtime->create_logical_region(ctx, bias_is, fs);
  LogicalRegion scale_lr = runtime->create_logical_region(ctx, bias_is, fs);
  LogicalRegion bias_grad_lr =
    runtime->create_logical_region(ctx, bias_grad_is, fs);
  LogicalRegion scale_grad_lr =
    runtime->create_logical_region(ctx, bias_grad_is, fs);
  IndexPartition bias_grad_ip =
    runtime->create_equal_partition(ctx, bias_grad_is, task_is);
  LogicalPartition bias_grad_lp =
    runtime->get_logical_partition(ctx, bias_grad_lr, bias_grad_ip);
  LogicalPartition scale_grad_lp =
    runtime->get_logical_partition(ctx, scale_grad_lr, bias_grad_ip);

  Tensor scale_tensor, bias_tensor;
  scale_tensor.region = scale_lr;
  scale_tensor.region_grad = scale_grad_lr;
  scale_tensor.part = LogicalPartition::NO_PART;
  scale_tensor.part_grad = scale_grad_lp;
  locals[0] = scale_tensor;
  bias_tensor.region = bias_lr;
  bias_tensor.region_grad = bias_grad_lr;
  bias_tensor.part = LogicalPartition::NO_PART;
  bias_tensor.part_grad = bias_grad_lp;
  locals[1] = bias_tensor;
  numLocals = 2;

  output = _input;
  output.region = output_lr;
  output.part = output_lp;
  output.region_grad = output_grad_lr;
  output.part_grad = output_grad_lp;
  printf("Create bn layer: output(%d %d %d %d)\n",
          output.adim[3], output.adim[2], output.adim[1], output.adim[0]);

  input_lps[0] = _input.part;
}

/*
  regions[0]: input
  regions[1]: output
  regions[2](I): scale
  regions[3](I): bias
*/
__host__
OpMeta* BatchNorm::init_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const BatchNorm* bm = (BatchNorm*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  const AccessorRO<float, 4> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 4> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 1> acc_scale(regions[2], FID_DATA);
  const AccessorRO<float, 1> acc_bias(regions[3], FID_DATA);
  Rect<1> rect_scale, rect_bias;
  Rect<4> rect_input, rect_output;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_scale = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_scale.accessor.is_dense_arbitrary(rect_scale));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  const float *scale_ptr = acc_scale.ptr(rect_scale.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);

  BatchNormMeta* m = new BatchNormMeta(handle);
#ifndef DISABLE_COMPUTATION
  m->relu = bm->relu;
  m->mode = HIPDNN_BATCHNORM_SPATIAL;
#if CUDNN_VERSION >= 7000
  m->mode = HIPDNN_BATCHNORM_SPATIAL_PERSISTENT;
#endif

  checkCUDNN(hipdnnCreateTensorDescriptor(&m->inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->biasTensor));

  assert(rect_input == rect_output);
  int input_w = rect_input.hi[0] - rect_input.lo[0] + 1;
  int input_h = rect_input.hi[1] - rect_input.lo[1] + 1;
  int channel = bm->inputs[0].pdim[2];
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        bm->inputs[0].pdim[3],
                                        channel, input_h, input_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        bm->inputs[0].pdim[3],
                                        channel, input_h, input_w));
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->biasTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        1, channel, 1, 1));
  //float *runningMean, *runningVar, *saveMean, *saveVar;
  checkCUDA(hipMalloc(&m->runningMean, sizeof(float) * channel));
  checkCUDA(hipMalloc(&m->runningVar, sizeof(float) * channel));
  checkCUDA(hipMalloc(&m->saveMean, sizeof(float) * channel));
  checkCUDA(hipMalloc(&m->saveVar, sizeof(float) * channel));
  if (m->relu) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0, 0.0, 0.0));
    
  }
#endif
  return m;
}

/*
  regions[0](O): scale, initilized to ones
  regions[1](O): bias, initilized to zeros
*/
__host__
void BatchNorm::init_para_task(const Task *task,
                               const std::vector<PhysicalRegion> &regions,
                               Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  //const BatchNorm* bm = (BatchNorm*) task->args;
  const AccessorWO<float, 1> acc_scale(regions[0], FID_DATA);
  const AccessorWO<float, 1> acc_bias(regions[1], FID_DATA);
  Rect<1> rect_scale, rect_bias;
  rect_scale = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  assert(acc_scale.accessor.is_dense_arbitrary(rect_scale));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float *scale_ptr = acc_scale.ptr(rect_scale.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  // init kernel and bias
#ifdef PARAMETER_ALL_ONES
  ones_kernel<<<GET_BLOCKS(rect_scale.volume()), CUDA_NUM_THREADS>>>(
      scale_ptr, rect_scale.volume());
  ones_kernel<<<GET_BLOCKS(rect_bias.volume()), CUDA_NUM_THREADS>>>(
      bias_ptr, rect_bias.volume());
#else
  //hipStream_t stream;
  //checkCUDA(hipStreamCreate(&stream));
  //hiprandGenerator_t genGPU;
  //hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_DEFAULT);
  //hiprandSetStream(genGPU, stream);
  //hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);
  //hiprandGenerateUniform(genGPU, scale_ptr, rect_scale.volume());
  assign_kernel<<<GET_BLOCKS(rect_scale.volume()), CUDA_NUM_THREADS>>>(
      scale_ptr, rect_scale.volume(), 1.0f);
  assign_kernel<<<GET_BLOCKS(rect_bias.volume()), CUDA_NUM_THREADS>>>(
      bias_ptr, rect_bias.volume(), 0.0f);
  //hiprandDestroyGenerator(genGPU);
#endif
}

__host__
void BatchNorm::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  // First we initialize the scale and bias parameters
  {
    TaskLauncher para_launcher(BATCHNORM_INIT_PARA_TASK_ID, TaskArgument(NULL, 0));
    para_launcher.add_region_requirement(
        RegionRequirement(locals[0].region, WRITE_DISCARD, EXCLUSIVE, locals[0].region));
    para_launcher.add_field(0, FID_DATA);
    para_launcher.add_region_requirement(
        RegionRequirement(locals[1].region, WRITE_DISCARD, EXCLUSIVE, locals[1].region));
    para_launcher.add_field(1, FID_DATA);
    runtime->execute_task(ctx, para_launcher);
  }
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher init_launcher(BATCHNORM_INIT_TASK_ID, task_is,
                              TaskArgument(this, sizeof(BatchNorm)), argmap);
  init_launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  init_launcher.add_field(1, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[0].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[0].region));
  init_launcher.add_field(2, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[1].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  init_launcher.add_field(3, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I): input
  regions[1](O): ouptut
  regions[2](I): scale
  regions[3](I): bias
*/
__host__
void BatchNorm::forward_task(const Task *task,
                             const std::vector<PhysicalRegion> &regions,
                             Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const BatchNorm* bm = (BatchNorm*) task->args;
  const BatchNormMeta* m = *((BatchNormMeta**) task->local_args);
  const AccessorRO<float, 4> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 4> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 1> acc_scale(regions[2], FID_DATA);
  const AccessorRO<float, 1> acc_bias(regions[3], FID_DATA);
  Rect<4> rect_input, rect_output;
  Rect<1> rect_scale, rect_bias;
  rect_input = runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_scale = runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_scale.accessor.is_dense_arbitrary(rect_scale));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  const float *scale_ptr = acc_scale.ptr(rect_scale.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);  

  hipEvent_t t_start, t_end;
  if (bm->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  coord_t numChannels = bm->inputs[0].pdim[2];
  assign_kernel<<<GET_BLOCKS(numChannels), CUDA_NUM_THREADS>>>(m->runningMean, numChannels, 0.0f);
  assign_kernel<<<GET_BLOCKS(numChannels), CUDA_NUM_THREADS>>>(m->runningVar, numChannels, 0.0f);
  checkCUDNN(hipdnnBatchNormalizationForwardTraining(
             m->handle.dnn, m->mode, &alpha, &beta, m->inputTensor, (void *)input_ptr,
             m->outputTensor, (void *)output_ptr, m->biasTensor, (void *)scale_ptr, (void *)bias_ptr,
             1.0, m->runningMean, m->runningVar, HIPDNN_BN_MIN_EPSILON,
             m->saveMean, m->saveVar));
  if (bm->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("BatchNorm forward time (BF) = %.2fms\n", elapsed);
  }
#endif
}

__host__
void BatchNorm::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(BATCHNORM_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(BatchNorm)), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[0].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[0].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[1].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  launcher.add_field(3, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](I): input
  regions[1](O): input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): scale
  regions[5](O): scale_grad
  regions[6](O): bias_grad
*/
__host__
void BatchNorm::backward_task(const Task *task,
                              const std::vector<PhysicalRegion> &regions,
                              Context ctx, Runtime *runtime)
{
#ifndef DISABLE_COMPUTATION
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  float alpha = 1.0f, beta = 0.0f;
  const BatchNorm* bm = (BatchNorm*) task->args;
  const BatchNormMeta* m = *((BatchNormMeta**) task->local_args);
  const AccessorRO<float, 4> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 4> acc_input_grad(regions[1], FID_DATA);
  const AccessorRO<float, 4> acc_output(regions[2], FID_DATA);
  const AccessorRW<float, 4> acc_output_grad(regions[3], FID_DATA);
  const AccessorRO<float, 1> acc_scale(regions[4], FID_DATA);
  const AccessorWO<float, 1> acc_scale_grad(regions[5], FID_DATA);
  const AccessorWO<float, 1> acc_bias_grad(regions[6], FID_DATA);
  Rect<4> rect_input, rect_input_grad, rect_output, rect_output_grad;
  Rect<1> rect_scale, rect_scale_grad, rect_bias_grad;
  rect_input =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_input_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_output =
    runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_output_grad =
    runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  rect_scale =
    runtime->get_index_space_domain(ctx, task->regions[4].region.get_index_space());
  rect_scale_grad =
    runtime->get_index_space_domain(ctx, task->regions[5].region.get_index_space());
  rect_bias_grad =
    runtime->get_index_space_domain(ctx, task->regions[6].region.get_index_space());
  // make sure all regions are dense
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_input_grad.accessor.is_dense_arbitrary(rect_input_grad));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_output_grad.accessor.is_dense_arbitrary(rect_output_grad));
  assert(acc_scale.accessor.is_dense_arbitrary(rect_scale));
  assert(acc_scale_grad.accessor.is_dense_arbitrary(rect_scale_grad));
  assert(acc_bias_grad.accessor.is_dense_arbitrary(rect_bias_grad));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *input_grad_ptr = acc_input_grad.ptr(rect_input_grad.lo);
  const float *output_ptr = acc_output.ptr(rect_output.lo);
  float *output_grad_ptr = acc_output_grad.ptr(rect_output_grad.lo);
  const float *scale_ptr = acc_scale.ptr(rect_scale.lo);
  float *scale_grad_ptr = acc_scale_grad.ptr(rect_scale_grad.lo);
  float *bias_grad_ptr = acc_bias_grad.ptr(rect_bias_grad.lo);

  hipEvent_t t_start, t_end;
  if (bm->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  if (m->relu) {
    int n = rect_output.volume();
    reluBackward<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(output_grad_ptr, output_ptr, n);
  }
  checkCUDNN(hipdnnBatchNormalizationBackward(
             m->handle.dnn, m->mode, &alpha, &beta, &alpha, &beta,
             m->inputTensor, input_ptr, m->outputTensor, output_grad_ptr,
             m->inputTensor, input_grad_ptr, m->biasTensor, scale_ptr,
             scale_grad_ptr, bias_grad_ptr, HIPDNN_BN_MIN_EPSILON,
             m->saveMean, m->saveVar));
  if (bm->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("BatchNorm backward time = %.2fms\n", elapsed);
  }
#endif
}

__host__
void BatchNorm::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }

  IndexLauncher launcher(BATCHNORM_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(BatchNorm)), argmap);
  // regions[0](I): input
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1](O): input_grad (we only need grad tensors)
  launcher.add_region_requirement(
      RegionRequirement(inputs[0].part_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(1, FID_DATA);
  // regions[2](I): output
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, output.region));
  launcher.add_field(2, FID_DATA);
  // regions[3](I/O): output_grad
  launcher.add_region_requirement(
      RegionRequirement(output.part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, output.region_grad));
  launcher.add_field(3, FID_DATA);
  // regions[4](I): filter
  launcher.add_region_requirement(
      RegionRequirement(locals[0].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[0].region));
  launcher.add_field(4, FID_DATA);
  // regions[5](O): filter_grad
  launcher.add_region_requirement(
      RegionRequirement(locals[0].part_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[0].region_grad));
  launcher.add_field(5, FID_DATA);
  // regions[6](O): bias_grad
  launcher.add_region_requirement(
      RegionRequirement(locals[1].part_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[1].region_grad));
  launcher.add_field(6, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
}

__host__
void BatchNorm::update(const FFModel& ff)
{
  //FIXME: we didn't sync batch norm parameters for now
}
