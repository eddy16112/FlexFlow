#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "model.h"
#include "cuda_helper.h"

Tensor FFModel::conv2d(std::string name,
                       Tensor input, int outChannels,
                       int kernelH, int kernelW,
                       int strideH, int strideW,
                       int paddingH, int paddingW, bool relu)
{
  printf("CP#1\n");
  assert(input.numDim == 4); /*NCHW*/
  int inChannels = input.adim[2];
  bool firstLayer = false;
  //if (input.region == inputImage.region)
  //  firstLayer = true;
  ParallelConfig pc;
  assert(config.find_parallel_config(name, pc));
  IndexSpaceT<4> task_is = IndexSpaceT<4>(get_or_create_task_is(pc));
  Conv2D *conv = new Conv2D(name, config, input, task_is,
                            inChannels, outChannels, kernelH, kernelW,
                            strideH, strideW, paddingH, paddingW,
                            relu, firstLayer);
  layers.push_back(conv);
  return conv->output;
}

/*
locals[0] = kernel
locals[1] = bias
*/
Conv2D::Conv2D(std::string _name, FFConfig _config,
               Tensor _input, IndexSpaceT<4> _task_is,
               int _in_channels, int _out_channels,
               int _kernel_h, int _kernel_w,
               int _stride_h, int _stride_w,
               int _padding_h, int _padding_w,
               bool _relu, bool _first_layer)
: Op(_name, _input), task_is(_task_is),
  in_channels(_in_channels), out_channels(_out_channels),
  kernel_h(_kernel_h), kernel_w(_kernel_w),
  stride_h(_stride_h), stride_w(_stride_w),
  padding_h(_padding_h), padding_w(_padding_w),
  relu(_relu), first_layer(_first_layer), profiling(_config.profiling)
{
  Context ctx = _config.lg_ctx;
  Runtime* runtime = _config.lg_hlr;
  Rect<4> part_rect = runtime->get_index_space_domain(ctx, task_is);
  num_replica = part_rect.volume();
  // Create output tensor
  int input_w = _input.adim[0];
  int input_h = _input.adim[1];
  int output_w = 1 + (input_w + 2 * padding_w - kernel_w) / stride_w;
  int output_h = 1 + (input_h + 2 * padding_h - kernel_h) / stride_h;
  int output_c = out_channels;
  int output_n = _input.adim[3];
  int num_par_w = part_rect.hi[0] - part_rect.lo[0] + 1;
  int num_par_h = part_rect.hi[1] - part_rect.lo[1] + 1;
  int num_par_c = part_rect.hi[2] - part_rect.lo[2] + 1;
  int num_par_n = part_rect.hi[3] - part_rect.lo[3] + 1;
 
  FieldSpace fs = _config.field_space;

  IndexSpaceT<4> output_is;
  {
    //const Legion::coord_t lo[4] = {0, 0, 0, 0};
    //const Legion::coord_t hi[4] = {output_w-1, output_h-1, output_c-1, output_n-1};
    Rect<4> output_rect(Point<4>(0, 0, 0, 0),
        Point<4>(output_w-1, output_h-1, output_c-1, output_n-1));
    output_is = runtime->create_index_space<4>(ctx, output_rect);
  }
  LogicalRegion output_lr = runtime->create_logical_region(ctx, output_is, fs);
  LogicalRegion output_grad_lr = runtime->create_logical_region(ctx, output_is, fs);
  int extent_w = (output_w + num_par_w - 1) / num_par_w;
  int extent_h = (output_h + num_par_h - 1) / num_par_h;
  int extent_c = output_c / num_par_c;
  int extent_n = output_n / num_par_n;
  assert(output_c % num_par_c == 0);
  assert(output_n % num_par_n == 0);
  Transform<4, 4, coord_t> transform;
  for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
      transform[i][j] = 0;
  transform[0][0] = extent_w;
  transform[1][1] = extent_h;
  transform[2][2] = extent_c;
  transform[3][3] = extent_n;
  IndexPartition output_ip;
  {
    //int lo[4] = {0, 0, 0, 0};
    //int hi[4] = {extent_w-1, extent_h-1, extent_c-1, extent_n-1};
    Rect<4> extent(Realm::Point<4>(0, 0, 0, 0),
        Realm::Point<4>(extent_w-1, extent_h-1, extent_c-1, extent_n-1));
    output_ip = runtime->create_partition_by_restriction(ctx, output_is, task_is, transform, extent);
    assert(runtime->is_index_partition_disjoint(ctx, output_ip));
    assert(runtime->is_index_partition_complete(ctx, output_ip));
  }
  LogicalPartition output_lp = runtime->get_logical_partition(ctx, output_lr, output_ip);
  LogicalPartition output_grad_lp =
    runtime->get_logical_partition(ctx, output_grad_lr, output_ip);

  int kernel_nc = num_replica * in_channels * out_channels;
  Rect<1, coord_t> kernel_rect(0, kernel_w * kernel_h * in_channels * out_channels - 1);
  Rect<1, coord_t> kernel_grad_rect(0, kernel_w * kernel_h * kernel_nc - 1);
  IndexSpaceT<1> kernel_is = runtime->create_index_space(ctx, kernel_rect);
  IndexSpaceT<1> kernel_grad_is = runtime->create_index_space(ctx, kernel_grad_rect);
  LogicalRegion kernel_lr = runtime->create_logical_region(ctx, kernel_is, fs);
  LogicalRegion kernel_grad_lr = runtime->create_logical_region(ctx, kernel_grad_is, fs);
  IndexPartition kernel_grad_ip =
    runtime->create_equal_partition(ctx, kernel_grad_is, task_is);
  LogicalPartition kernel_grad_lp =
    runtime->get_logical_partition(ctx, kernel_grad_lr, kernel_grad_ip);
  Tensor kernel_tensor;
  kernel_tensor.numDim = 0;
  kernel_tensor.region = kernel_lr;
  kernel_tensor.region_grad = kernel_grad_lr;
  kernel_tensor.part = LogicalPartition::NO_PART;
  kernel_tensor.part_grad = kernel_grad_lp;
  locals[0] = kernel_tensor;

  int bias_nc = num_replica * out_channels;
  Rect<1, coord_t> bias_grad_rect(0, bias_nc - 1);
  Rect<1, coord_t> bias_rect(0, out_channels - 1);
  IndexSpaceT<1> bias_is = runtime->create_index_space(ctx, bias_rect);
  IndexSpaceT<1> bias_grad_is = runtime->create_index_space(ctx, bias_grad_rect);
  LogicalRegion bias_lr = runtime->create_logical_region(ctx, bias_is, fs);
  LogicalRegion bias_grad_lr =
    runtime->create_logical_region(ctx, bias_grad_is, fs);
  IndexPartition bias_grad_ip =
    runtime->create_equal_partition(ctx, bias_grad_is, task_is);
  LogicalPartition bias_grad_lp =
    runtime->get_logical_partition(ctx, bias_grad_lr, bias_grad_ip);
  Tensor bias_tensor;
  bias_tensor.numDim = 0;
  bias_tensor.region = bias_lr;
  bias_tensor.region_grad = bias_grad_lr;
  bias_tensor.part = LogicalPartition::NO_PART;
  bias_tensor.part_grad = bias_grad_lp;
  locals[1] = bias_tensor;
  numLocals = 2;

  output.numDim = 4;
  output.adim[0] = output_w;
  output.adim[1] = output_h;
  output.adim[2] = out_channels;
  output.adim[3] = _input.adim[3];
  output.pdim[0] = extent_w;
  output.pdim[1] = extent_h;
  output.pdim[2] = extent_c;
  output.pdim[3] = extent_n;
  output.region = output_lr;
  output.part = output_lp;
  output.region_grad = output_grad_lr;
  output.part_grad = output_grad_lp;
  printf("Create conv layer: output(n=%d c=%d h=%d w=%d)\n",
         output.adim[3], output.adim[2], output.adim[1], output.adim[0]);

  // Compute partition bound for input
  Rect<4> input_part_rect =
    runtime->get_index_partition_color_space(ctx, inputs[0].part.get_index_partition());
  if (input_part_rect == part_rect) {
    input_lps[0] = _input.part;
  } else {
    printf("WARNING: input has a different partition!!!\n");
    IndexSpaceT<4> input_is = IndexSpaceT<4>(inputs[0].region.get_index_space());
    //extent_w = stride_w * (output.pdim[0]-1) + kernel_w - 2 * padding_w;
    //extent_h = stride_h * (output.pdim[1]-1) + kernel_h - 2 * padding_h;
    //extent_nc = inputs[0].adim[2] * inputs[0].adim[3] / num_par_n;
    extent_w = (inputs[0].adim[0] + num_par_w - 1) / num_par_w;
    extent_h = (inputs[0].adim[1] + num_par_h - 1) / num_par_h;
    extent_c = inputs[0].adim[2] / num_par_c;
    extent_n = inputs[0].adim[3] / num_par_n;
    assert(inputs[0].adim[2] % num_par_c == 0);
    assert(inputs[0].adim[3] % num_par_n == 0);
    //transform[0][0] = stride_w * output.pdim[0];
    //transform[1][1] = stride_h * output.pdim[1];
    //transform[2][2] = extent_nc;
    transform[0][0] = extent_w;
    transform[1][1] = extent_h;
    transform[2][2] = extent_c;
    transform[3][3] = extent_n;

    IndexPartition input_ip;
    {
      //int lo[4] = {0, 0, 0, 0};
      //int hi[4] = {extent_w-1, extent_h-1, extent_c-1, extent_n-1};
      Rect<4> extent_i(Realm::Point<4>(0, 0, 0, 0),
          Realm::Point<4>(extent_w-1, extent_h-1, extent_c-1, extent_n-1));
      input_ip = runtime->create_partition_by_restriction(ctx,
          input_is, task_is, transform, extent_i);
      assert(runtime->is_index_partition_disjoint(ctx, input_ip));
      assert(runtime->is_index_partition_complete(ctx, input_ip));
    }
    input_lps[0] = runtime->get_logical_partition(ctx, inputs[0].region, input_ip);
  }
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y);
hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw);
hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx);
/*
  regions[0]: input
  regions[1]: output
  regions[2](I): filter
  regions[3](I): bias
*/
__host__
OpMeta* Conv2D::init_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Conv2D* conv = (Conv2D*) task->args;
  FFHandler handle = *((const FFHandler*) task->local_args);
  const AccessorRO<float, 3> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 3> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 1> acc_filter(regions[2], FID_DATA);
  const AccessorRO<float, 1> acc_bias(regions[3], FID_DATA);
  Rect<1> rect_filter, rect_bias;
  Rect<3> rect_input, rect_output;
  rect_input = runtime->get_index_space_domain(
                   ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(
                    ctx, task->regions[1].region.get_index_space());
  rect_filter = runtime->get_index_space_domain(
                    ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(
                  ctx, task->regions[3].region.get_index_space());
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  const float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);

  Conv2DMeta* m = new Conv2DMeta(handle);
  m->relu = conv->relu;
  m->first_layer = conv->first_layer;
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&m->outputTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&m->filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&m->convDesc));

  int input_w = rect_input.hi[0] - rect_input.lo[0] + 1;
  int input_h = rect_input.hi[1] - rect_input.lo[1] + 1;
  int output_w = rect_output.hi[0] - rect_output.lo[0] + 1;
  int output_h = rect_output.hi[1] - rect_output.lo[1] + 1;
  printf("init conv (input): n(%d) c(%d) h(%d) w(%d)\n", conv->inputs[0].pdim[3],
         conv->inputs[0].pdim[2], input_h, input_w);
  printf("init conv (output): n(%d) c_out(%d) h(%d) w(%d)\n", conv->output.pdim[3],
         conv->output.pdim[2], output_h, output_w);
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->inputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        conv->inputs[0].pdim[3],
                                        conv->inputs[0].pdim[2],
                                        input_h,
                                        input_w));
  
  checkCUDNN(hipdnnSetTensor4dDescriptor(m->biasTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        1,
                                        conv->output.pdim[2],
                                        1,
                                        1));

  printf("filterDim: kernel(%d %d) c_out(%d)\n", conv->kernel_h, conv->kernel_w, conv->output.pdim[2]);
  checkCUDNN(hipdnnSetFilter4dDescriptor(m->filterDesc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        conv->output.pdim[2],
                                        conv->inputs[0].pdim[2],
                                        conv->kernel_h,
                                        conv->kernel_w));

  //printf("convDim: padding(%d %d) stride(%d %d)\n", conv->padding_h, conv->padding_w, conv->stride_h, conv->stride_w);
  int pad_h = ((output_h - 1) * conv->stride_h + conv->kernel_h - input_h + 1) / 2;
  int pad_w = ((output_w - 1) * conv->stride_w + conv->kernel_w - input_w + 1) / 2;
  if (pad_h != conv->padding_h)
    printf("Warning: changing conv_padding_h to satisfy output_h size\n");
  if (pad_w != conv->padding_w)
    printf("Warning: changing conv_padding_w to satisfy output_w size\n");

  checkCUDNN(hipdnnSetConvolution2dDescriptor(m->convDesc,
                                             pad_h,//conv->padding_h,
                                             pad_w,//conv->padding_w,
                                             conv->stride_h,
                                             conv->stride_w,
                                             1/*upscale_x*/,
                                             1/*upscale_y*/,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));

  int n, c, h, w;
  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(m->convDesc,
                                                   m->inputTensor,
                                                   m->filterDesc,
                                                   &n, &c, &h, &w));
  assert(n == conv->output.pdim[3]);
  assert(c == conv->output.pdim[2]);
  assert(h == output_h);
  assert(w == output_w);

  checkCUDNN(hipdnnSetTensor4dDescriptor(m->outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n, c, h, w));
  // select forward algorithm
  m->fwdAlgo = selectConvolutionForwardAlgorithm(m->handle.dnn, m->inputTensor, input_ptr,
                                                 m->filterDesc, filter_ptr, m->convDesc,
                                                 m->handle.workSpace, m->handle.workSpaceSize,
                                                 m->outputTensor, output_ptr);
  // select backward filter algorithm
  m->bwdFilterAlgo = selectConvolutionBackwardFilterAlgorithm(
                         m->handle.dnn, m->inputTensor, input_ptr,
                         m->outputTensor, output_ptr,
                         m->convDesc, m->handle.workSpace, m->handle.workSpaceSize,
                         m->filterDesc, (void*)filter_ptr);
  // select backward data algorithm
  m->bwdDataAlgo = selectConvolutionBackwardDataAlgorithm(
                       m->handle.dnn, m->filterDesc, filter_ptr,
                       m->outputTensor, output_ptr,
                       m->convDesc, m->handle.workSpace, m->handle.workSpaceSize,
                       m->inputTensor, (void*)input_ptr);
  if (m->relu) {
    checkCUDNN(hipdnnCreateActivationDescriptor(&m->actiDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(m->actiDesc, HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN, 0.0, 0.0, 0.0));
  }
  return m;
}

/*
  regions[0](O): filter
  regions[1](O): bias
*/
__host__
void Conv2D::init_para_task(const Task *task,
                            const std::vector<PhysicalRegion> &regions,
                            Context ctx, Runtime *runtime)
{
  assert(regions.size() == 2);
  assert(task->regions.size() == 2);
  const Conv2D* conv = (Conv2D*) task->args;
  const AccessorWO<float, 1> acc_filter(regions[0], FID_DATA);
  const AccessorWO<float, 1> acc_bias(regions[1], FID_DATA);
  Rect<1> rect_filter, rect_bias;
  rect_filter = runtime->get_index_space_domain(
                    ctx, task->regions[0].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(
                  ctx, task->regions[1].region.get_index_space());
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  // init kernel and bias
#ifdef PARAMETER_ALL_ONES
  coord_t filter_elements = conv->inputs[0].adim[2] * conv->output.adim[2] * conv->kernel_h * conv->kernel_w;
  ones_kernel<<<GET_BLOCKS(filter_elements), CUDA_NUM_THREADS>>>(
      filter_ptr, filter_elements);
  ones_kernel<<<GET_BLOCKS(filter_elements), CUDA_NUM_THREADS>>>(
      bias_ptr, conv->output.pdim[2]);
#else
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  hiprandGenerator_t genGPU;
  hiprandCreateGenerator(&genGPU, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetStream(genGPU, stream);
  hiprandSetPseudoRandomGeneratorSeed(genGPU, 1234ULL);
  coord_t filter_elements = conv->inputs[0].adim[2] * conv->output.adim[2] 
                          * conv->kernel_h * conv->kernel_w;
  float factor = 1.0f / sqrt(filter_elements / conv->output.adim[2]);
  printf("factor = %.4f elements = %d\n", factor, filter_elements / conv->output.adim[2]);
  assert(filter_elements == (coord_t) rect_filter.volume());
  hiprandGenerateUniform(genGPU, filter_ptr, filter_elements);
  scale_kernel<<<GET_BLOCKS(filter_elements), CUDA_NUM_THREADS>>>(
      filter_ptr, filter_elements, -factor, factor);
  hiprandGenerateUniform(genGPU, bias_ptr, conv->output.pdim[2]);
  scale_kernel<<<GET_BLOCKS(conv->output.pdim[2]), CUDA_NUM_THREADS>>>(
      bias_ptr, conv->output.pdim[2], -factor, factor);
  hiprandDestroyGenerator(genGPU);
#endif
}

__host__
void Conv2D::init(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;

  // First we initialize the filter and bias parameters
  {
    TaskLauncher para_launcher(CONV2D_INIT_PARA_TASK_ID, TaskArgument(this, sizeof(Conv2D)));
    para_launcher.add_region_requirement(
        RegionRequirement(locals[0].region, WRITE_DISCARD, EXCLUSIVE, locals[0].region));
    para_launcher.add_field(0, FID_DATA);
    para_launcher.add_region_requirement(
        RegionRequirement(locals[1].region, WRITE_DISCARD, EXCLUSIVE, locals[1].region));
    para_launcher.add_field(1, FID_DATA);
    runtime->execute_task(ctx, para_launcher);
  }

  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    FFHandler handle = ff.handlers[idx++];
    argmap.set_point(*it, TaskArgument(&handle, sizeof(FFHandler)));
  }
  IndexLauncher init_launcher(CONV2D_INIT_TASK_ID, task_is,
                              TaskArgument(this, sizeof(Conv2D)), argmap);
  init_launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  init_launcher.add_field(0, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  init_launcher.add_field(1, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[0].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[0].region));
  init_launcher.add_field(2, FID_DATA);
  init_launcher.add_region_requirement(
      RegionRequirement(locals[1].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  init_launcher.add_field(3, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, init_launcher);
  fm.wait_all_results();
  idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    meta[idx++] = fm.get_result<OpMeta*>(*it);
  }
}

/*
  regions[0](I): input
  regions[1](O): output
  regions[2](I): filter
  regions[3](I): bias
*/
__host__
void Conv2D::forward_task(const Task *task,
                          const std::vector<PhysicalRegion> &regions,
                          Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  float alpha = 1.0f, beta = 0.0f;
  const Conv2D* conv = (Conv2D*) task->args;
  const Conv2DMeta* m = *((Conv2DMeta**) task->local_args);
  const AccessorRO<float, 4> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 4> acc_output(regions[1], FID_DATA);
  const AccessorRO<float, 1> acc_filter(regions[2], FID_DATA);
  const AccessorRO<float, 1> acc_bias(regions[3], FID_DATA);
  Rect<4> rect_input, rect_output;
  Rect<1> rect_filter, rect_bias;
  rect_input = runtime->get_index_space_domain(
                   ctx, task->regions[0].region.get_index_space());
  rect_output = runtime->get_index_space_domain(
                    ctx, task->regions[1].region.get_index_space());
  rect_filter = runtime->get_index_space_domain(
                    ctx, task->regions[2].region.get_index_space());
  rect_bias = runtime->get_index_space_domain(
                  ctx, task->regions[3].region.get_index_space());
  //for (int i = 0; i < 3; i++) printf("rect_input.hi = %lld lo = %lld\n", rect_input.hi[i], rect_input.lo[i]);
  //for (int i = 0; i < 3; i++) printf("rect_output.hi = %lld lo = %lld\n", rect_output.hi[i], rect_output.lo[i]);
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *output_ptr = acc_output.ptr(rect_output.lo);
  const float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  const float *bias_ptr = acc_bias.ptr(rect_bias.lo);  

  //printf("fwdAlgo(%d), bwdFilterALgo(%d), bwdDataAlgo(%d)\n", (int)m->fwdAlgo,(int) m->bwdFilterAlgo,(int) m->bwdDataAlgo);
  hipEvent_t t_start, t_end;
  if (conv->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  checkCUDNN(hipdnnConvolutionForward(m->handle.dnn, &alpha,
                                     m->inputTensor, input_ptr,
                                     m->filterDesc, filter_ptr,
                                     m->convDesc, m->fwdAlgo,
                                     m->handle.workSpace, m->handle.workSpaceSize,
                                     &beta, m->outputTensor, output_ptr));

  checkCUDNN(hipdnnAddTensor(m->handle.dnn, &alpha, m->biasTensor,
                            bias_ptr, &alpha, m->outputTensor, output_ptr));
  if (m->relu) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn, m->actiDesc,
                                      &alpha, m->outputTensor, output_ptr,
                                      &beta, m->outputTensor, output_ptr));
  }
  if (conv->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Conv2D forward time (CF) = %.2fms\n", elapsed);
  }
}

__host__
void Conv2D::forward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }
  IndexLauncher launcher(CONV2D_FWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Conv2D)), argmap);
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, output.region));
  launcher.add_field(1, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[0].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[0].region));
  launcher.add_field(2, FID_DATA);
  launcher.add_region_requirement(
      RegionRequirement(locals[1].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[1].region));
  launcher.add_field(3, FID_DATA);

  runtime->execute_index_space(ctx, launcher);
}

/*
  regions[0](I): input
  regions[1](O): input_grad
  regions[2](I): output
  regions[3](I/O): output_grad
  regions[4](I): filter
  regions[5](O): filter_grad
  regions[6](O): bias_grad
*/
__host__
void Conv2D::backward_task(const Task *task,
                           const std::vector<PhysicalRegion> &regions,
                           Context ctx, Runtime *runtime)
{
  assert(regions.size() == 7);
  assert(task->regions.size() == 7);
  float alpha = 1.0f, beta = 0.0f;
  const Conv2D* conv = (Conv2D*) task->args;
  const Conv2DMeta* m = *((Conv2DMeta**) task->local_args);
  const AccessorRO<float, 4> acc_input(regions[0], FID_DATA);
  const AccessorWO<float, 4> acc_input_grad(regions[1], FID_DATA);
  const AccessorRO<float, 4> acc_output(regions[2], FID_DATA);
  const AccessorRW<float, 4> acc_output_grad(regions[3], FID_DATA);
  const AccessorRO<float, 1> acc_kernel(regions[4], FID_DATA);
  const AccessorWO<float, 1> acc_kernel_grad(regions[5], FID_DATA);
  const AccessorWO<float, 1> acc_bias_grad(regions[6], FID_DATA);
  Rect<4> rect_input, rect_input_grad, rect_output, rect_output_grad;
  Rect<1> rect_kernel, rect_kernel_grad, rect_bias_grad;
  rect_input =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_input_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_output =
    runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_output_grad =
    runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  rect_kernel =
    runtime->get_index_space_domain(ctx, task->regions[4].region.get_index_space());
  rect_kernel_grad =
    runtime->get_index_space_domain(ctx, task->regions[5].region.get_index_space());
  rect_bias_grad =
    runtime->get_index_space_domain(ctx, task->regions[6].region.get_index_space());
  // make sure all regions are dense
  assert(acc_input.accessor.is_dense_arbitrary(rect_input));
  assert(acc_input_grad.accessor.is_dense_arbitrary(rect_input_grad));
  assert(acc_output.accessor.is_dense_arbitrary(rect_output));
  assert(acc_output_grad.accessor.is_dense_arbitrary(rect_output_grad));
  assert(acc_kernel.accessor.is_dense_arbitrary(rect_kernel));
  assert(acc_kernel_grad.accessor.is_dense_arbitrary(rect_kernel_grad));
  assert(acc_bias_grad.accessor.is_dense_arbitrary(rect_bias_grad));
  const float *input_ptr = acc_input.ptr(rect_input.lo);
  float *input_grad_ptr = acc_input_grad.ptr(rect_input_grad.lo);
  const float *output_ptr = acc_output.ptr(rect_output.lo);
  float *output_grad_ptr = acc_output_grad.ptr(rect_output_grad.lo);
  const float *kernel_ptr = acc_kernel.ptr(rect_kernel.lo);
  float *kernel_grad_ptr = acc_kernel_grad.ptr(rect_kernel_grad.lo);
  float *bias_grad_ptr = acc_bias_grad.ptr(rect_bias_grad.lo);

  hipEvent_t t_start, t_end;
  if (conv->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start);
  }
  hipStream_t stream;
  checkCUDA(hipStreamCreate(&stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  if (m->relu) {
    int n = rect_output.volume();
    reluBackward<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(output_grad_ptr, output_ptr, n);
  }
  // Compute filter gradiant
  checkCUDNN(hipdnnConvolutionBackwardFilter(m->handle.dnn, &alpha,
                                            m->inputTensor, input_ptr,
                                            m->outputTensor, output_grad_ptr,
                                            m->convDesc, m->bwdFilterAlgo,
                                            m->handle.workSpace, m->handle.workSpaceSize,
                                            &beta, m->filterDesc, kernel_grad_ptr));
  // Compute bias gradiant
  checkCUDNN(hipdnnConvolutionBackwardBias(m->handle.dnn, &alpha,
                                          m->outputTensor, output_grad_ptr,
                                          &beta, m->biasTensor, bias_grad_ptr));
  // no need to compute input_grad if we are the first layer
  if (!m->first_layer) {
    // Compute data gradiant
    checkCUDNN(hipdnnConvolutionBackwardData(m->handle.dnn, &alpha,
                                            m->filterDesc, kernel_ptr,
                                            m->outputTensor, output_grad_ptr,
                                            m->convDesc, m->bwdDataAlgo,
                                            m->handle.workSpace, m->handle.workSpaceSize,
                                            &beta, m->inputTensor, input_grad_ptr));
  }
  if (conv->profiling) {
    hipEventRecord(t_end);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("Conv2D backward time = %.2fms\n", elapsed);
  }
}

__host__
void Conv2D::backward(const FFModel& ff)
{
  ArgumentMap argmap;
  Context ctx = ff.config.lg_ctx;
  Runtime* runtime = ff.config.lg_hlr;
  Rect<4> rect = runtime->get_index_space_domain(ctx, task_is);
  int idx = 0;
  for (PointInRectIterator<4> it(rect); it(); it++) {
    OpMeta* mp = meta[idx++];
    argmap.set_point(*it, TaskArgument(&mp, sizeof(OpMeta*)));
  }

  IndexLauncher launcher(CONV2D_BWD_TASK_ID, task_is,
                         TaskArgument(this, sizeof(Conv2D)), argmap);
  // regions[0](I): input
  launcher.add_region_requirement(
      RegionRequirement(input_lps[0], 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, inputs[0].region));
  launcher.add_field(0, FID_DATA);
  // regions[1](O): input_grad (we only need grad tensors)
  launcher.add_region_requirement(
      RegionRequirement(inputs[0].part_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, inputs[0].region_grad));
  launcher.add_field(1, FID_DATA);
  // regions[2](I): output
  launcher.add_region_requirement(
      RegionRequirement(output.part, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, output.region));
  launcher.add_field(2, FID_DATA);
  // regions[3](I/O): output_grad
  launcher.add_region_requirement(
      RegionRequirement(output.part_grad, 0/*projection id*/,
                        READ_WRITE, EXCLUSIVE, output.region_grad));
  launcher.add_field(3, FID_DATA);
  // regions[4](I): filter
  launcher.add_region_requirement(
      RegionRequirement(locals[0].region, 0/*projection id*/,
                        READ_ONLY, EXCLUSIVE, locals[0].region));
  launcher.add_field(4, FID_DATA);
  // regions[5](O): filter_grad
  launcher.add_region_requirement(
      RegionRequirement(locals[0].part_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[0].region_grad));
  launcher.add_field(5, FID_DATA);
  // regions[6](O): bias_grad
  launcher.add_region_requirement(
      RegionRequirement(locals[1].part_grad, 0/*projection id*/,
                        WRITE_DISCARD, EXCLUSIVE, locals[1].region_grad));
  launcher.add_field(6, FID_DATA);
  FutureMap fm = runtime->execute_index_space(ctx, launcher);
  // TODO: remove this line
  //if (first_layer)
    //fm.wait_all_results();
}

/*
  regions[0](I/O): filter
  regions[1](I): filter_grad
  regions[2](I/O): bias
  regions[3](I): bias_grad
*/
__host__
void Conv2D::update_task(const Task *task,
                         const std::vector<PhysicalRegion> &regions,
                         Context ctx, Runtime *runtime)
{
  assert(regions.size() == 4);
  assert(task->regions.size() == 4);
  const Conv2D* conv = (Conv2D*) task->args;
  const AccessorRW<float, 1> acc_filter(regions[0], FID_DATA);
  const AccessorRO<float, 1> acc_filter_grad(regions[1], FID_DATA);
  const AccessorRW<float, 1> acc_bias(regions[2], FID_DATA);
  const AccessorRO<float, 1> acc_bias_grad(regions[3], FID_DATA);
  Rect<1> rect_filter, rect_filter_grad, rect_bias, rect_bias_grad;
  rect_filter =
    runtime->get_index_space_domain(ctx, task->regions[0].region.get_index_space());
  rect_filter_grad =
    runtime->get_index_space_domain(ctx, task->regions[1].region.get_index_space());
  rect_bias =
    runtime->get_index_space_domain(ctx, task->regions[2].region.get_index_space());
  rect_bias_grad =
    runtime->get_index_space_domain(ctx, task->regions[3].region.get_index_space());
  size_t filter_size = rect_filter.volume();
  size_t bias_size = rect_bias.volume();
  assert(filter_size == conv->in_channels * conv->out_channels
                        * conv->kernel_w * conv->kernel_h);
  assert(bias_size == conv->out_channels);
  assert(filter_size * conv->num_replica == rect_filter_grad.volume());
  assert(bias_size * conv->num_replica == rect_bias_grad.volume());
  assert(acc_filter.accessor.is_dense_arbitrary(rect_filter));
  assert(acc_filter_grad.accessor.is_dense_arbitrary(rect_filter_grad));
  assert(acc_bias.accessor.is_dense_arbitrary(rect_bias));
  assert(acc_bias_grad.accessor.is_dense_arbitrary(rect_bias_grad));
  float *filter_ptr = acc_filter.ptr(rect_filter.lo);
  const float *filter_grad_ptr = acc_filter_grad.ptr(rect_filter_grad.lo);
  float *bias_ptr = acc_bias.ptr(rect_bias.lo);
  const float *bias_grad_ptr = acc_bias_grad.ptr(rect_bias_grad.lo);
  updateGAS(filter_ptr, filter_grad_ptr, filter_size,
            conv->num_replica, conv->learning_rate);
  updateGAS(bias_ptr, bias_grad_ptr, bias_size,
            conv->num_replica, conv->learning_rate);
}

__host__
void Conv2D::update(const FFModel& ff)
{
  // Synchronize the learning rate
  learning_rate = ff.config.learningRate;
  assert(num_replica > 0);
  // Only aggregate parameters if more than one replica
  if (num_replica > 1) {
    Context ctx = ff.config.lg_ctx;
    Runtime* runtime = ff.config.lg_hlr;
    TaskLauncher launcher(CONV2D_UPD_TASK_ID, TaskArgument(this, sizeof(Conv2D)));
    launcher.add_region_requirement(
      RegionRequirement(locals[0].region, READ_WRITE, EXCLUSIVE, locals[0].region));
    launcher.add_field(0, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[0].region_grad, READ_ONLY, EXCLUSIVE, locals[0].region_grad));
    launcher.add_field(1, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[1].region, READ_WRITE, EXCLUSIVE, locals[1].region));
    launcher.add_field(2, FID_DATA);
    launcher.add_region_requirement(
      RegionRequirement(locals[1].region_grad, READ_ONLY, EXCLUSIVE, locals[1].region_grad));
    launcher.add_field(3, FID_DATA);
    runtime->execute_task(ctx, launcher);
  }
}

hipdnnConvolutionFwdAlgo_t
selectConvolutionForwardAlgorithm(hipdnnHandle_t handle,
                                  const hipdnnTensorDescriptor_t xDesc, const void* x,
                                  const hipdnnFilterDescriptor_t wDesc, const void* w,
                                  const hipdnnConvolutionDescriptor_t convDesc,
                                  void* workSpace, size_t workSpaceSize,
                                  const hipdnnTensorDescriptor_t yDesc, void* y)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionFwdAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionForwardAlgorithmEx(
      handle, xDesc, x, wDesc, w, convDesc, yDesc, y,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("forwardAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdFilterAlgo_t
selectConvolutionBackwardFilterAlgorithm(hipdnnHandle_t handle,
                                         const hipdnnTensorDescriptor_t xDesc, const void* x,
                                         const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                         const hipdnnConvolutionDescriptor_t convDesc,
                                         void* workSpace, size_t workSpaceSize,
                                         const hipdnnFilterDescriptor_t dwDesc, void* dw)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdFilterAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithmEx(
      handle, xDesc, x, dyDesc, dy, convDesc, dwDesc, dw,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdFilterAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}

hipdnnConvolutionBwdDataAlgo_t
selectConvolutionBackwardDataAlgorithm(hipdnnHandle_t handle,
                                       const hipdnnFilterDescriptor_t wDesc, const void* w,
                                       const hipdnnTensorDescriptor_t dyDesc, const void* dy,
                                       const hipdnnConvolutionDescriptor_t convDesc,
                                       void* workSpace, size_t workSpaceSize,
                                       const hipdnnTensorDescriptor_t dxDesc, void* dx)
{
  const int reqAlgCnt = 8;
  int cnt = 0;
  hipdnnConvolutionBwdDataAlgoPerf_t perfResults[reqAlgCnt];
  checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithmEx(
      handle, wDesc, w, dyDesc, dy, convDesc, dxDesc, dx,
      reqAlgCnt, &cnt, perfResults, workSpace, workSpaceSize));
  assert(cnt > 0);
  checkCUDNN(perfResults[0].status);
  printf("bwdDataAlgo(%d) time(%.2lf)\n", perfResults[0].algo, perfResults[0].time);
  return perfResults[0].algo;
}
